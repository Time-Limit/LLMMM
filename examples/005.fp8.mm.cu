#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <hipblas.h>
#include <cuda/barrier>
#include <hip/hip_fp8.h>
#include <cuda_pipeline_primitives.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <random>
#include <vector>

#include "util/macro.h"
#include "util/util.cuh"

using namespace LLMMM;

constexpr int limit = 128;

__global__ void fp32_naive_mm(const float* A, const float* B, float* C, int M, int N, int K)
{
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  int m = blockIdx.y * blockDim.y + threadIdx.y;

  if (n >= N || m >= M) {
    return;
  }

  A += m * K;
  B += n;
  double sum = 0.0;
#pragma unroll
  for (int k = 0; k < K; ++k) {
    sum += A[k] * B[k * N];
  }
  C[m * N + n] = sum;
}

void launch_fp32_naive_mm(const float* A, const float* B, float* C, int M, int N, int K)
{
  dim3 threads_per_block(16, 16);
  dim3 blocks_per_grid((N + threads_per_block.x - 1) / threads_per_block.x,
                       (M + threads_per_block.y - 1) / threads_per_block.y);

  fp32_naive_mm<<<blocks_per_grid, threads_per_block>>>(A, B, C, M, N, K);
  CHECK_CUDA_ERROR();
}

template<int BLOCK_M, int BLOCK_N, int WARP_M, int WARP_N, int LOOP_K, bool C_SCALE_TRANSPOSE>
__global__ void fp8_gemm_blockwise_quant_A_1x128__B_128x128__C_1x128(const __hip_fp8_e4m3_fnuz* A,
                                                                     const float*         A_scale_transposed,
                                                                     const __hip_fp8_e4m3_fnuz* B,
                                                                     const float*         B_scale,
                                                                     half*                C,
                                                                     float*               C_scale,
                                                                     int                  M,
                                                                     int                  N,
                                                                     int                  K)
{
  //   static_assert(BLOCK_N == 128 && LOOP_K == 128);
  //
  //   constexpr int M_WARP_COUNT = BLOCK_M / WARP_M;
  //   constexpr int N_WARP_COUNT = BLOCK_N / WARP_N;
  //   constexpr int WARP_COUNT   = M_WARP_COUNT * N_WARP_COUNT;
  //   constexpr int THREAD_COUNT = WARP_COUNT * 32;
  //
  //   const int warp_id = threadIdx.x / 32;
  //   const int lane_id = threadIdx.x % 32;
  //   const int m_warp_id = warp_id % M_WARP_COUNT;
  //   const int n_warp_id = warp_id / N_WARP_COUNT;
  //
  //   if  (m_warp_id && n_warp_id) {}
  //
  //   // for mma_m16n8k32
  //   const int m_block_offset = blockIdx.y * BLOCK_M;
  //   const int n_block_offset = blockIdx.x * BLOCK_N;
  //   const int m_warp_offset  = warp_id / M_WARP_COUNT * WARP_M;
  //   const int n_warp_offset  = warp_id % M_WARP_COUNT * WARP_N;
  //   constexpr int M_GROUP_COUNT_PER_WARP = WARP_M / 8;
  //   constexpr int N_GROUP_COUNT_PER_WARP = WARP_N / 16;
  //   union {
  //     uint16_t      ldm[2][2];
  //     __hip_fp8_e4m3_fnuz mma[8];
  //   } A_mma_reg;
  //   union {
  //     __hip_fp8_e4m3_fnuz ldm[2][8];
  //     __hip_fp8_e4m3_fnuz mma[16];
  //   } B_mma_reg;
  //   float C_mma_reg[M_GROUP_COUNT_PER_WARP][N_GROUP_COUNT_PER_WARP][4] = {0};
  //
  //   // for LDG
  //   __shared__ __hip_fp8_e4m3_fnuz A_sm[LOOP_K / 32][BLOCK_M][32];
  //   __shared__ __hip_fp8_e4m3_fnuz B_sm[BLOCK_N / 16][LOOP_K][16];
  //   __shared__ float         A_scale_sm[BLOCK_M];
  //   __shared__ float         B_scale_sm;
  //   float                    A_scale_reg[4];
  //   float                    B_scale_reg;
  //   static_assert(BLOCK_M * LOOP_K % THREAD_COUNT == 0);
  //   constexpr int A_LDG_DATA_PER_THREAD = BLOCK_M * LOOP_K / THREAD_COUNT;
  //   static_assert(8 <= A_LDG_DATA_PER_THREAD && (A_LDG_DATA_PER_THREAD & (A_LDG_DATA_PER_THREAD - 1)) == 0);
  //   constexpr int A_BYTES_PER_LDG = A_LDG_DATA_PER_THREAD >= 16 ? 16 : A_LDG_DATA_PER_THREAD;
  //   static_assert(A_BYTES_PER_LDG == 8 || A_BYTES_PER_LDG == 16);
  //   constexpr int A_LDG_LOOP_COUNT    = A_LDG_DATA_PER_THREAD / A_BYTES_PER_LDG;
  //   __hip_fp8_e4m3_fnuz A_ldg_reg[A_LDG_LOOP_COUNT][A_BYTES_PER_LDG];
  //   constexpr int B_LDG_DATA_PER_THREAD = BLOCK_N * LOOP_K / THREAD_COUNT;
  //   constexpr int B_BYTES_PER_LDG = 16;
  //   static_assert(B_LDG_DATA_PER_THREAD % B_BYTES_PER_LDG == 0);
  //   constexpr int B_LDG_LOOP_COUNT    = B_LDG_DATA_PER_THREAD / B_BYTES_PER_LDG;
  //   // This check is to ensure that matrix B can be transposed via registers.
  //   static_assert(B_LDG_LOOP_COUNT % 8 == 0);
  //   __hip_fp8_e4m3_fnuz B_ldg_reg[B_LDG_LOOP_COUNT][B_BYTES_PER_LDG];
  //
  //   for (int k_loop_offset = 0; k_loop_offset < K; k_loop_offset += LOOP_K) {
  //     for (int loop = 0; loop < A_LDG_LOOP_COUNT; ++loop) {
  //       const int group    = (loop * THREAD_COUNT + threadIdx.x) * A_BYTES_PER_LDG / 32;
  //       const int m_global = m_block_offset + group % BLOCK_M;
  //       const int k_global = k_loop_offset + group / BLOCK_M * 32 + threadIdx.x * A_BYTES_PER_LDG % 32;
  //       if constexpr (A_BYTES_PER_LDG == 8) {
  //         FETCH_FLOAT2(A_ldg_reg[loop][0], A[OFFSET(m_global, k_global, K)]);
  //       }
  //       if constexpr (A_BYTES_PER_LDG == 16) {
  //         FETCH_FLOAT4(A_ldg_reg[loop][0], A[OFFSET(m_global, k_global, K)]);
  //       }
  //     }
  //     for (int loop = 0; loop < A_LDG_LOOP_COUNT; ++loop) {
  //       const int group = (loop * THREAD_COUNT + threadIdx.x) * A_BYTES_PER_LDG / 32;
  //       const int m_sm  = group % BLOCK_M;
  //       const int k_sm  = group / BLOCK_M * 32 + threadIdx.x * A_BYTES_PER_LDG % 32;
  //       if constexpr (A_BYTES_PER_LDG == 8) {
  //         STORE_FLOAT2(A_sm[k_sm / 32][m_sm][k_sm % 32], A_ldg_reg[loop]);
  //       }
  //       if constexpr (A_BYTES_PER_LDG == 16) {
  //         STORE_FLOAT4(A_sm[k_sm / 32][m_sm][k_sm % 32], A_ldg_reg[loop]);
  //       }
  //     }
  //     for (int loop = 0; loop < B_LDG_LOOP_COUNT; ++loop) {
  //       const int group    = (loop * THREAD_COUNT + threadIdx.x) * B_BYTES_PER_LDG / 16;
  //       const int k_global = k_loop_offset + group % LOOP_K;
  //       const int n_global = n_block_offset + group / LOOP_K * 16 + threadIdx.x * B_BYTES_PER_LDG % 16;
  //       FETCH_FLOAT4(B_ldg_reg[loop], B[OFFSET(k_global, n_global, N)]);
  //     }
  //     for (int loop = 0; loop < B_LDG_LOOP_COUNT; ++loop) {
  //       const int group = (loop * THREAD_COUNT + threadIdx.x) * B_BYTES_PER_LDG / 16;
  //       const int k_sm  = group % LOOP_K;
  //       const int n_sm  = group / LOOP_K * 16 + threadIdx.x * B_BYTES_PER_LDG % 16;
  //       STORE_FLOAT4(B_sm[n_sm / 16][k_sm][n_sm % 16], B_ldg_reg[loop]);
  //     }
  //     if (warp_id == 0) {
  //       FETCH_FLOAT(B_scale_reg, B_scale[OFFSET(k_loop_offset / 128, n_block_offset / 128, N / 128)]);
  //       STORE_FLOAT(B_scale_sm, B_scale_reg);
  //       static_assert(BLOCK_M <= 128);
  //       const int m = lane_id * 4 % BLOCK_M;
  //       FETCH_FLOAT4(A_scale_reg, A_scale_transposed[OFFSET(k_loop_offset / 128, m, M)]);
  //       STORE_FLOAT4(A_scale_sm[m], A_scale_reg);
  //     }
  //     __syncthreads();
  //   }
  //   {
  //     using T     = half;
  //     union {
  //       T _2x4[2][4];
  //       T _1x8[8];
  //     } C_transposed[M_GROUP_COUNT_PER_WARP][N_GROUP_COUNT_PER_WARP / 2];
  //     T* C_ptr =
  //       &C[(m_block_offset + m_warp_offset + lane_id / 4) * N + n_block_offset + n_warp_offset + lane_id % 4 * 8];
  //     for (int mg = 0; mg < M_GROUP_COUNT_PER_WARP; ++mg) {
  //       for (int ng = 0; ng < N_GROUP_COUNT_PER_WARP; ++ng) {
  //         T casted[4] = {C_mma_reg[mg][ng][0], C_mma_reg[mg][ng][1], C_mma_reg[mg][ng][2], C_mma_reg[mg][ng][3]};
  //         asm volatile("movmatrix.sync.aligned.m8n8.trans.b16 %0, %1;\n"
  //                      : "=r"(*(uint32_t*)&C_transposed[mg][ng / 2]._2x4[ng % 2][0])
  //                      : "r"(*(uint32_t*)&casted[0]));
  //         asm volatile("movmatrix.sync.aligned.m8n8.trans.b16 %0, %1;\n"
  //                      : "=r"(*(uint32_t*)&C_transposed[mg][ng / 2]._2x4[ng % 2][2])
  //                      : "r"(*(uint32_t*)&casted[2]));
  //         shfl_23_and_01(C_transposed[mg][ng / 2]._2x4[ng % 2], 0x1, lane_id);
  //         if ((ng + 1) % 2 == 0) {
  //           shfl_4567_and_0123(C_transposed[mg][ng / 2]._1x8, 0x2, lane_id);
  //           asm volatile("st.global.wt.v4.f32 [%0], {%1, %2, %3, %4};"
  //                        :
  //                        : "l"(C_ptr + mg * 8 * N + (ng - 1) * 16),
  //                          "f"(*(const float*)&C_transposed[mg][ng / 2]._1x8[0]),
  //                          "f"(*(const float*)&C_transposed[mg][ng / 2]._1x8[2]),
  //                          "f"(*(const float*)&C_transposed[mg][ng / 2]._1x8[4]),
  //                          "f"(*(const float*)&C_transposed[mg][ng / 2]._1x8[6])
  //                        : "memory");
  //         }
  //       }
  //     }
  //   }
}

template<bool C_SCALE_TRANSPOSE>
void fp8_gemm_blockwise_quant_A_1x128__B_128x128__C_1x128(const __hip_fp8_e4m3_fnuz* A,
                                                          const float*         A_scale_transposed,
                                                          const __hip_fp8_e4m3_fnuz* B,
                                                          const float*         B_scale,
                                                          half*                C,
                                                          float*               C_scale,
                                                          int                  M,
                                                          int                  N,
                                                          int                  K,
                                                          hipStream_t         stream)
{
  constexpr int BLOCK_M = 128;
  constexpr int BLOCK_N = 128;
  constexpr int LOOP_K  = 128;
  constexpr int WARP_M  = 64;
  constexpr int WARP_N  = 64;
  static_assert(BLOCK_M % WARP_M == 0);
  static_assert(BLOCK_N % WARP_N == 0);
  static_assert(WARP_M % 16 == 0);
  static_assert(WARP_N % 8 == 0);
  static_assert(LOOP_K == 128);
  constexpr int WARP_COUNT = BLOCK_M / WARP_M * BLOCK_N / WARP_N;
  static_assert(0 < WARP_COUNT && WARP_COUNT <= 4 && (WARP_COUNT & (WARP_COUNT - 1)) == 0);
  if (!(M % BLOCK_M == 0 && N % BLOCK_N == 0 && K % LOOP_K == 0)) {
    throw std::runtime_error("M or N or K are not aligned.");
  }
  dim3 grid(N / BLOCK_N, M / BLOCK_M);
  dim3 block(WARP_COUNT * 32);
  fp8_gemm_blockwise_quant_A_1x128__B_128x128__C_1x128<BLOCK_M, BLOCK_N, WARP_M, WARP_N, LOOP_K, C_SCALE_TRANSPOSE>
    <<<grid, block, 0, stream>>>(A, A_scale_transposed, B, B_scale, C, C_scale, M, N, K);
}

template<int BLOCK_M, int BLOCK_N, int QUANT_M, int QUANT_N, bool SCALE_TRANPOSE>
__global__ void fp8_blockwise_symmetric_quantization(const float* x, __hip_fp8_e4m3_fnuz* q, float* scale, int M, int N)
{
  const int m_block_offset = BLOCK_M * blockIdx.y;
  const int n_block_offset = BLOCK_N * blockIdx.x;
  const int warp_id        = threadIdx.x / 32;
  const int lane_id        = threadIdx.x % 32;

  constexpr int THREAD_COUNT = 128;
  constexpr int WARP_COUNT   = THREAD_COUNT / 32;

  constexpr int DATA_PER_THREAD = BLOCK_M * BLOCK_N / THREAD_COUNT;
  static_assert(DATA_PER_THREAD == 128);
  constexpr int LOOP_COUNT = DATA_PER_THREAD / 4;

  const float fp8_e4m3_range = 448;

  const float max_float = INT_MIN;

  float max = max_float;

  const int scale_N = (N + QUANT_N - 1) / QUANT_N;

  __shared__ float x_sm[QUANT_M * QUANT_N];

  for (int loop = 0; loop < LOOP_COUNT; ++loop) {
    int m = m_block_offset + loop * WARP_COUNT + warp_id;
    if (m < M) {
      float data[4];
      FETCH_FLOAT4_PREFETCH_256B_WITH_SRC_PTR(data, &x[OFFSET(m, n_block_offset + lane_id * 4, N)]);
      if (QUANT_M == 128) {
        STORE_FLOAT4(x_sm[OFFSET(loop * WARP_COUNT + warp_id, lane_id * 4, BLOCK_N)], data);
      }
      max = (max < fabs(data[0])) ? fabs(data[0]) : max;
      max = (max < fabs(data[1])) ? fabs(data[1]) : max;
      max = (max < fabs(data[2])) ? fabs(data[2]) : max;
      max = (max < fabs(data[3])) ? fabs(data[3]) : max;
      if constexpr (QUANT_M == 1) {
        max     = warp_reduce_max(max);
        max     = warp_broadcast(0, max);
        float s = max / fp8_e4m3_range;

        __hip_fp8_e4m3_fnuz quanted[4] = {
          __hip_fp8_e4m3_fnuz(data[0] / s),
          __hip_fp8_e4m3_fnuz(data[1] / s),
          __hip_fp8_e4m3_fnuz(data[2] / s),
          __hip_fp8_e4m3_fnuz(data[3] / s),
        };

        max = max_float;

        STORE_FLOAT(q[OFFSET(m, n_block_offset + lane_id * 4, N)], quanted);
        static_assert(BLOCK_N == QUANT_N);
        if (lane_id == 0) {
          if constexpr (SCALE_TRANPOSE) {
            STORE_FLOAT(scale[OFFSET(n_block_offset / QUANT_N, m, M)], s);
          } else {
          STORE_FLOAT(scale[OFFSET(m, n_block_offset / QUANT_N, scale_N)], s);
          }
        }
      }
    }
  }
  if constexpr (QUANT_M == 128) {
    __shared__ float block_max_value[WARP_COUNT];
    max = warp_reduce_max(max);
    if (lane_id == 0) {
      static_assert(WARP_COUNT == 4);
      block_max_value[warp_id] = max;
    }
    __syncthreads();
    float max4[4];
    FETCH_FLOAT4(max4[0], block_max_value[0]);
    max = (max4[0] > max4[1]) ? max4[0] : max4[1];
    max = (max > max4[2]) ? max : max4[2];
    max = (max > max4[3]) ? max : max4[3];
    float s = max / fp8_e4m3_range;
    for (int loop = 0; loop < LOOP_COUNT; ++loop) {
      int m = m_block_offset + loop * WARP_COUNT + warp_id;
      if (m < M) {
        float data[4];
        FETCH_FLOAT4(data[0], x_sm[OFFSET(loop * WARP_COUNT + warp_id, lane_id * 4, BLOCK_N)]);

        __hip_fp8_e4m3_fnuz quanted[4] = {
          __hip_fp8_e4m3_fnuz(data[0] / s),
          __hip_fp8_e4m3_fnuz(data[1] / s),
          __hip_fp8_e4m3_fnuz(data[2] / s),
          __hip_fp8_e4m3_fnuz(data[3] / s),
        };
        STORE_FLOAT(q[OFFSET(m_block_offset + loop * WARP_COUNT + warp_id, n_block_offset + lane_id * 4, N)], quanted);
      }
      static_assert(BLOCK_N == QUANT_N);
      if (threadIdx.x == 0) {
        if constexpr (SCALE_TRANPOSE) {
          STORE_FLOAT(
            scale[OFFSET(n_block_offset / QUANT_N, (m_block_offset + loop * WARP_COUNT + warp_id) / QUANT_M, M)], s);
        } else {
          STORE_FLOAT(
            scale[OFFSET((m_block_offset + loop * WARP_COUNT + warp_id) / QUANT_M, n_block_offset / QUANT_N, scale_N)],
            s);
        }
      }
    }
  }
}

template<int QUANT_M, int QUANT_N, bool SCALE_TRANPOSE>
void fp8_blockwise_symmetric_quantization(
  const float* x, __hip_fp8_e4m3_fnuz* q, float* scale, int M, int N, hipStream_t stream)
{
  static_assert(QUANT_M == 1 || QUANT_M == 128);
  static_assert(QUANT_N == 128);
  constexpr int BLOCK_M = 128;
  constexpr int BLOCK_N = 128;

  dim3 grid((N + BLOCK_N - 1) / BLOCK_N, (M + BLOCK_M - 1) / BLOCK_M);
  dim3 block(128);
  auto kSmemSize   = 0;
  auto kernel_func = &fp8_blockwise_symmetric_quantization<BLOCK_M, BLOCK_N, QUANT_M, QUANT_N, SCALE_TRANPOSE>;
  CHECK_CUDA_RETURN(hipFuncSetAttribute(reinterpret_cast<const void*>(kernel_func), hipFuncAttributeMaxDynamicSharedMemorySize, kSmemSize));
  fp8_blockwise_symmetric_quantization<BLOCK_M, BLOCK_N, QUANT_M, QUANT_N, SCALE_TRANPOSE>
    <<<grid, block, 0, stream>>>(x, q, scale, M, N);
}

int main()
{
  static const int M = (1 << 12), N = (1 << 12), K = (1 << 12);

  std::vector<float>                    h_A(M * K), h_B(K * N), h_C(M * N);
  std::random_device                    rd;
  std::mt19937                          gen(rd());
  std::uniform_real_distribution<float> dis(-5, 5);
  for (auto& vec : {&h_A, &h_B}) {
#if 1
    for (auto& data : *vec) {
      data = dis(gen);
    }
#else
    if (vec == &h_A) {
      for (size_t i = 0; i < vec->size(); ++i) {
        int row            = i / K;
        int col            = i % K;
        vec->operator[](i) = (row == col);
        if (row < limit && col < limit) {
          vec->operator[](i) = row * limit + col;
        }
        else {
          vec->operator[](i) = 0;
        }
      }
    }
    if (vec == &h_B) {
      for (size_t i = 0; i < vec->size(); ++i) {
        int row = i / N;
        int col = i % N;
        if (row < limit && col < limit) {
          vec->operator[](i) = row * limit + col;
        }
        else {
          vec->operator[](i) = 0;
        }
      }
    }
#endif
  }

  float *d_A, *d_B, *d_C;
  for (auto& pair : {std::make_pair(h_A, &d_A), std::make_pair(h_B, &d_B), std::make_pair(h_C, &d_C)}) {
    const std::vector<float>& h      = pair.first;
    float*&                   device = *pair.second;
    CHECK_CUDA_RETURN(hipMalloc(&device, sizeof(float) * h.size()));
    CHECK_CUDA_RETURN(hipMemcpy(device, h.data(), sizeof(float) * h.size(), hipMemcpyDefault));
  }

  {
    hipMemset(d_C, 0, M * N * sizeof(float));
    launch_fp32_naive_mm(d_A, d_B, d_C, M, N, K);
    hipMemcpy(h_C.data(), d_C, sizeof(float) * h_C.size(), hipMemcpyDefault);
    CHECK_CUDA_ERROR();
  }

  std::vector<float>         h_A_s(M * K / 128);
  float*                     d_A_s;
  std::vector<__hip_fp8_e4m3_fnuz> h_A_q(M * K);
  __hip_fp8_e4m3_fnuz*             d_A_q;
  std::vector<float>         h_B_s(N / 128 * K / 128);
  float*                     d_B_s;
  std::vector<__hip_fp8_e4m3_fnuz> h_B_q(N * K);
  __hip_fp8_e4m3_fnuz*             d_B_q;
  std::vector<float>         h_C_s(M * N / 128);
  float*                     d_C_s;
  std::vector<__hip_fp8_e4m3_fnuz> h_C_q(M * N);
  __hip_fp8_e4m3_fnuz*             d_C_q;
  for (auto& pair : {std::make_pair(h_A_s, &d_A_s), std::make_pair(h_B_s, &d_B_s), std::make_pair(h_C_s, &d_C_s)}) {
    const std::vector<float>& h      = pair.first;
    float*&                   device = *pair.second;
    CHECK_CUDA_RETURN(hipMalloc(&device, sizeof(float) * h.size()));
  }

  for (auto& pair : {std::make_pair(h_A_q, &d_A_q), std::make_pair(h_B_q, &d_B_q), std::make_pair(h_C_q, &d_C_q)}) {
    const std::vector<__hip_fp8_e4m3_fnuz>& h      = pair.first;
    __hip_fp8_e4m3_fnuz*&                   device = *pair.second;
    CHECK_CUDA_RETURN(hipMalloc(&device, sizeof(__hip_fp8_e4m3_fnuz) * h.size()));
  }

  fp8_blockwise_symmetric_quantization<1, 128, true>(d_A, d_A_q, d_A_s, M, N, nullptr);
  CHECK_CUDA_ERROR();

  fp8_blockwise_symmetric_quantization<128, 128, false>(d_B, d_B_q, d_B_s, N, K, nullptr);
  CHECK_CUDA_ERROR();

  half*             d_C_fp16;
  std::vector<half> h_C_fp16(M * N);
  CHECK_CUDA_RETURN(hipMalloc(&d_C_fp16, sizeof(half) * h_C_fp16.size()));
  fp8_gemm_blockwise_quant_A_1x128__B_128x128__C_1x128<true>(
    d_A_q, d_A_s, d_B_q, d_B_s, d_C_fp16, d_C_s, M, N, K, nullptr);
  CHECK_CUDA_ERROR();

  CHECK_CUDA_RETURN(hipFree(d_A));
  CHECK_CUDA_RETURN(hipFree(d_B));
  CHECK_CUDA_RETURN(hipFree(d_C));
  CHECK_CUDA_RETURN(hipFree(d_A_s));
  CHECK_CUDA_RETURN(hipFree(d_B_s));
  CHECK_CUDA_RETURN(hipFree(d_C_s));
  CHECK_CUDA_RETURN(hipFree(d_A_q));
  CHECK_CUDA_RETURN(hipFree(d_B_q));
  CHECK_CUDA_RETURN(hipFree(d_C_q));
  CHECK_CUDA_RETURN(hipFree(d_C_fp16));
  return 0;
}
